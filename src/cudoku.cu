
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <getopt.h>
#include <string>
#include <iostream>
#include <vector>

#include <algorithm>
#include <fstream>
#include <cassert>
#include <iomanip>
#include <chrono>
#include <string>
#include <vector>

#include <unistd.h>
#include <omp.h>

const int NUM_BLOCKS = 20000;
__device__ bool solution_found;
__device__ int solution_idx;

// both parameters are num_blocks size arrays
__global__ void solveBoard(char* boards, int* statuses, int board_size){
    __shared__ int progress_flag;
    __shared__ int done_flag;
    __shared__ int error_flag;
    __shared__ int min_possibility_count;
    __shared__ int min_possibility_thread_idx_x;
    __shared__ int min_possibility_thread_idx_y;

    char* board = &boards[sizeof(char) * board_size * board_size * blockIdx.x];
    int status = statuses[sizeof(char) * blockIdx.x];

    int board_dim = board_size * board_size;
    int inner_board_dim = sqrtf(board_size);

    // status = 0 if idle, 1 if running, 2 if done?
    if(status == 1){
        if(threadIdx.x < board_size && threadIdx.y < board_size){
            int possibilities_count = 0;
            char possibles = 0; // a bitmask for 1-board_size all possible

            // First thread in each block should reset the reductions.
            if( threadIdx.x == 0 && threadIdx.y == 0){
                progress_flag = 1;
                error_flag = 0;
                min_possibility_count = board_size;
                min_possibility_thread_idx_x = board_size;
                min_possibility_thread_idx_y = board_size;
            }

            while(progress_flag){
                if(threadIdx.x == 0 && threadIdx.y == 0){
                    progress_flag = 0;
                    done_flag = 1;
                }
                __syncthreads();

                // Get cell value and check if it has been filled
                int board_value = board[threadIdx.x * board_size + threadIdx.y];
                if(board_value){
                    break;
                }

                done_flag = 0;
                possibles = 0;
                for(int i = 0; i < board_size; ++i){
                    // Check the current row: (threadIdx.x, i)
                    int row_value = board[threadIdx.x * board_size + i];
                    if(row_value){
                        possibles |= (1 << (row_value-1));
                    }

                    // Current column: (i, threadIdx.y)
                    int col_value = board[i * board_size + threadIdx.y];
                    if(col_value){
                        possibles |= (1 << (col_value-1));
                    }

                }

                int inner_board_x = threadIdx.x - ( threadIdx.x % inner_board_dim);
                int inner_board_y = threadIdx.y - ( threadIdx.y % inner_board_dim);

                // check 3x3 subboard
                for(int i = inner_board_x; i < inner_board_x + inner_board_dim; ++i ){
                    for(int j = inner_board_y; j < inner_board_y + inner_board_dim; ++j ){
                        int inner_board_value = board[i * board_size + j];
                        if(inner_board_value){
                            possibles |= 1 << (inner_board_value-1);
                        }
                    }
                }

                // Find Deterministic updates first
                possibilities_count = 0;
                int temp = possibles;
                int update = 0;
                for(int i = 0; i < board_size; ++i){
                    if(!(temp & 1)){
                        possibilities_count += 1;
                        update = i + 1;
                    }
                    temp >>= 1;
                }

                // Deterministic Progress can be made
                if(possibilities_count == 1){
                    board[threadIdx.x * board_size + threadIdx.y] = update;
                    progress_flag = 1;
                }

                // If unfilled cell has no possibilities, then error
                if(possibilities_count == 0){
                    error_flag = 1;
                }
                __syncthreads();
            }

            // Flag is set only when every cell has been filled
            if(done_flag){
                solution_found = true;
                solution_idx = sizeof(char) * board_dim * blockIdx.x;
                return;
            }

            // If error flag is set, set status to idle
            if(error_flag){
                statuses[sizeof(char) * blockIdx.x] = 0;
                return;
            }

            // No Deterministic Progress can be made in any cell.
            // First, find cell with minimum number of possibilities
            if(possibilities_count != 0){
                atomicMin(&min_possibility_count, possibilities_count);
            }
            __syncthreads();

            if(possibilities_count == min_possibility_count){
                atomicMin(&min_possibility_thread_idx_x, threadIdx.x);
            }

            if(possibilities_count == min_possibility_count){
                atomicMin(&min_possibility_thread_idx_y, threadIdx.y);
            }
            __syncthreads();

            // Fork on possibilities of cell with mininum possibilities
            if(threadIdx.x == min_possibility_thread_idx_x && threadIdx.y == min_possibility_thread_idx_y){
                int next_block_index = blockIdx.x;

                for(int i = 0; i < board_size; ++i){
                    if(!(possibles & 1)){
                        int possible_value = i + 1;

                        if(next_block_index != blockIdx.x){
                            // next_block == 0 ? 1 : 0, i.e. atomic compare a block to 0 (idle) and set to 1 (working)
                            while(next_block_index < NUM_BLOCKS && atomicCAS(statuses + next_block_index, 0, 1) == 0)
                                next_block_index++;
                        }

                        if(next_block_index <= NUM_BLOCKS){
                            printf("Scheduling %d, %d to take a value of %d\n", threadIdx.x, threadIdx.y, possible_value);
                            char* new_board = &boards[sizeof(char) * board_dim * next_block_index];
                            memcpy(new_board, board, sizeof(char) * board_dim);
                            new_board[ threadIdx.x * board_size + threadIdx.y] = possible_value;
                        }
                        else{
                            break;
                        }
                    }
                    possibles >>= 1;
                }
            }
        }
    }
}

int solveBoardHost(std::vector<char> first_board){

    int board_size = first_board.size();
    char* boards;
    int* statuses;
    int status = 1;


    hipMalloc(&boards, sizeof(char) * board_size * NUM_BLOCKS);
    hipMalloc(&statuses, sizeof(int) * NUM_BLOCKS);

    hipMemcpy(boards, first_board.data(), sizeof(char) * board_size, hipMemcpyHostToDevice);
    hipMemcpy(statuses, &status, sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(9,9);
    dim3 gridDim(NUM_BLOCKS);

    solveBoard<<<gridDim, blockDim>>>(boards, statuses, 9);

    hipDeviceSynchronize();

}
