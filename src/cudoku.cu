#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <getopt.h>
#include <string>
#include <iostream>
#include <vector>

#include <algorithm>
#include <fstream>
#include <cassert>
#include <iomanip>
#include <chrono>
#include <string>
#include <vector>

#include <unistd.h>
#include <omp.h>

#include "cudoku.h"

const int NUM_BLOCKS = 50000;


__global__ void solveBoard(char* boards, int* statuses, int board_size, int* solution_idx, int* solution_found){
    
    //.Static shared memory for block constants
    __shared__ int progress_flag;
    __shared__ int done_flag;
    __shared__ int error_flag;
    __shared__ int min_possibility_count;
    __shared__ int min_possibility_thread_idx;

    // Dynamic shared memory for possibility sets based on board size
    extern __shared__ int dynamic_shared_mem[];

    int* row_possibles = dynamic_shared_mem;
    int* col_possibles = &dynamic_shared_mem[board_size];
    int* inner_possibles = &col_possibles[board_size];

    char* board = &boards[board_size * board_size * blockIdx.x];
    int status = statuses[blockIdx.x];

    // Various size consants
    int board_dim = board_size * board_size;
    int inner_board_size = sqrtf(board_size);
    int inner_row = threadIdx.x / inner_board_size;
    int inner_col = threadIdx.y / inner_board_size;
    int inner_idx = inner_row * inner_board_size + inner_col;
    

    // status = 0 if idle, 1 if running, 2 if done?
    if(status == 1 && threadIdx.x < board_size && threadIdx.y < board_size){
        int possible_ct = 0;
        int possibles = 0; // a bitmask for 1-board_size all possible
        // char update = 0;

        // First thread in each block should reset the reductions.
        if( threadIdx.x == 0 && threadIdx.y == 0){
            progress_flag = 1;
            error_flag = 0;
            done_flag = 0;
            min_possibility_count = board_size;
            min_possibility_thread_idx = board_size*board_size;
        }
        __syncthreads();

        // Loop while progress can be made
        while(progress_flag && !done_flag && !error_flag){

            if( threadIdx.x == 0 && threadIdx.y == 0){
                progress_flag = false;
                error_flag = false;
                done_flag = true;
            }

            if(threadIdx.x == 0 and threadIdx.y < board_size){
                row_possibles[threadIdx.y] = 0;
                col_possibles[threadIdx.y] = 0;
                inner_possibles[threadIdx.y] = 0;
            }
            possible_ct = 0;
            __syncthreads();
            
            // Get cell value and check if it has been filled
            int val = board[ threadIdx.x * board_size + threadIdx.y]; 
            int mask = 1 << (val-1);

            // Generate row, column, inner board possibilities cooperatively.
            // Use atomic updates to check if conflicting updates are made (and we need to fail).
            if(val){
                int old;
                old = atomicOr(&row_possibles[threadIdx.x], mask);

                if(old & mask){
                    atomicExch(&error_flag, 1);
                }

                old = atomicOr(&col_possibles[threadIdx.y], mask);
                if((old & mask)){
                    error_flag = true;
                }

                old = atomicOr(&inner_possibles[inner_idx], mask);
                if((old >> (val - 1)) & 1){
                    error_flag = true;
                }
            }
            __syncthreads();

            // Update deterministically if possible
            if(!val){
                
                done_flag = false;

                possibles = row_possibles[ threadIdx.x ];
                possibles |= col_possibles[ threadIdx.y ];
                possibles |= inner_possibles[inner_idx];

                int last_possible = 0;
                possible_ct = 0;

                for(int possible = 1; possible < board_size + 1; ++possible){
                    if(!(possibles & (1 << (possible - 1)))){
                        last_possible = possible;
                        possible_ct += 1;
                    }
                }
                // No possible values --> this solution is wrong somewhere.
                if(possible_ct == 0)
                    error_flag = true;
                
                // One possible value --> deterministic update.
                if(possible_ct == 1){
                    board[threadIdx.x * board_size + threadIdx.y] = last_possible;
                    progress_flag = true;
                }
            }
            __syncthreads();
        }

        // If error flag is set, set status to idle for rescheduling
        if(error_flag){
            if( threadIdx.x + threadIdx.y == 0)
                statuses[blockIdx.x] = 0;
            return;
        }

        // Flag is set only when every cell has been filled
        if(done_flag){
            if(threadIdx.x + threadIdx.y == 0){
                *solution_found = true;
                *solution_idx = blockIdx.x * board_dim;
            }
            return;
        }

        // No Deterministic Progress can be made in any cell.
        // First, find cell with minimum number of possibilities
        if(possible_ct != 0){
            atomicMin(&min_possibility_count, possible_ct);
        }
        __syncthreads();

        // Then find minimum cell index of those to update (arbitrary but fixed choice)
        if(possible_ct == min_possibility_count){
            atomicMin(&min_possibility_thread_idx, threadIdx.x * board_size + threadIdx.y);
        }

        __syncthreads();

        // Fork on possibilities of cell with mininum possibilities
        if(min_possibility_thread_idx == ( threadIdx.x * board_size + threadIdx.y)){
            int next_block_index = blockIdx.x;

            for(int possible = 1; possible < board_size+1; ++possible){
                if(!(possibles & (1 << (possible - 1)))){

                    if(next_block_index != blockIdx.x){
                        
                        // next_block == 0 ? 1 : 0, i.e. atomic compare a block to 0 (idle) and set to 1 (working)
                        while(next_block_index < NUM_BLOCKS && atomicCAS(&statuses[next_block_index], 0, 1) == 1)
                            next_block_index++;
                    }

                    if(next_block_index < NUM_BLOCKS){
                        char* new_board = &boards[next_block_index * board_size * board_size];
                        memcpy(new_board, board, sizeof(char) * board_dim);
                        new_board[ threadIdx.x * board_size + threadIdx.y] = possible;

                        next_block_index++;
                    }
                }
            }
        }
    }
}

std::vector<char> solveBoardHost(std::vector<char> board){

    int board_size = board.size();
    char* boards;
    int* statuses;
    int status = 1;

    int solution_found = 0;
    int* solution_found_device;
    int solution_idx;
    int* solution_idx_device;

    // Allocate and initialize global memory
    hipMalloc(&boards, sizeof(char) * board_size * NUM_BLOCKS);
    hipMalloc(&statuses, sizeof(int) * NUM_BLOCKS);
    hipMalloc(&solution_found_device, sizeof(int));
    hipMalloc(&solution_idx_device, sizeof(int));
    
    hipMemset(statuses, 0, sizeof(int) * NUM_BLOCKS);
    hipMemcpy(statuses, &status, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(boards, board.data(), sizeof(char) * board_size, hipMemcpyHostToDevice);
    hipMemcpy(solution_found_device, &solution_found, sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(9,9);
    dim3 gridDim(NUM_BLOCKS);

    int shared_memory_req = sizeof(int) * sqrt(board_size) * 3;

    // Call kernel in a loop to reschedule blocks until one finds a solution
    while(!solution_found){
        solveBoard<<<gridDim, blockDim, shared_memory_req>>>(
            boards, statuses, 9, solution_idx_device, solution_found_device
        );
        // hipDeviceSynchronize();
        hipMemcpy(&solution_found, solution_found_device, sizeof(int), hipMemcpyDeviceToHost);
    }
    
    // Copy board data back to host
    hipMemcpy(&solution_idx, solution_idx_device, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(board.data(), boards + solution_idx, board_size, hipMemcpyDeviceToHost);

    return board;
}
